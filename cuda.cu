#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define I 5000
#define J 2

#define BLOCKSIZEx 512

/*-Global variables-*/
struct timeval startwtime, endwtime;
double seq_time;

/**---Host function declarations---**/
int Blocks(int x, int b){ return ((x % b) != 0) ? (x / b + 1) : (x / b); }
void getData(double *, double *);									// get dataset
void printTable(double *);

/***-----Device function declarations-----***/
__device__ void sort(double *, int);									// Insertion sort
__device__ double KNN(double *, int, int);								// KNN algorithm for updating the bandwidth
__device__ double gaussian(double distance, double bandwidth){
  return exp(-distance / (2 * pow(bandwidth, 2)));
}
__global__ void kernel(int *, int *, double *, double *, double *);

/****-------------Main programm-------------****/
int main(int argc, char** argv)
{
  int i, l, k = 0, intsize = sizeof(int), doubsize = sizeof(double);					// k = neighbors for KNN algorithm
  int conv = 0;
  double *d_x, *d_y, *d_m;
  double *x, *y, *m;
  int *d_c, *d_k;
  while ((k < 1 || k > I - 1) && (conv <= 0))
  {
    printf("Give number of neighbors( > 1 & < elements-1), used for calculating bandwidth :\n");
    scanf("%d", &k);
    printf("And also the number of iterations for convergence (> 0): \n");
    scanf("%d", &conv);
  }
  x = (double *)malloc((I * J) * doubsize);
  y = (double *)malloc((I * J) * doubsize);
  m = (double *)malloc((I * J) * doubsize);

  getData(x, y);
  //printTable(x);
  printf("\n");

  hipMalloc(&d_c, intsize);
  hipMalloc(&d_k, intsize);
  hipMalloc(&d_x, (I*J)*doubsize);
  hipMalloc(&d_y, (I*J)*doubsize);
  hipMalloc(&d_m, (I*J)*doubsize);
  hipMemcpy(d_c, &conv, intsize, hipMemcpyHostToDevice);
  hipMemcpy(d_k, &k, intsize, hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, (I*J)*doubsize, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, (I*J)*doubsize, hipMemcpyHostToDevice);
  hipMemcpy(d_m, m, (I*J)*doubsize, hipMemcpyHostToDevice);

  dim3 gridSize(Blocks(I*J, BLOCKSIZEx));
  dim3 blockSize(BLOCKSIZEx);

  gettimeofday (&startwtime, NULL);

  kernel<<<gridSize,blockSize>>>(d_c, d_k, d_x, d_y, d_m);
  hipDeviceSynchronize();

  gettimeofday (&endwtime, NULL);
  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
	+ endwtime.tv_sec - startwtime.tv_sec);

  hipMemcpy(m, d_m, (I*J)*doubsize, hipMemcpyDeviceToHost);
  for(i = 0; i < I; i++){
    printf("\nMean for [%d] element is : ", i);
    for(l = 0; l < J; l++) printf("%lf ", m[i*J+l]);
  }

  printf("\n\nKernel clock time = %f\n", seq_time);

  free(x); free(y);
  hipFree(d_x); hipFree(d_y); hipFree(d_m);

  return 0;
}


/*****---------Host functions---------*****/
void getData(double *x, double *y) {
  int i, j;
  FILE *file = fopen("dataset.txt", "r");
  for (i = 0; i < I; i++) {
    for (j = 0; j < J; j++)
    {
      fscanf(file, "%lf", &x[i*J+j]);
      y[i*J+j] = x[i*J+j];
    }
  }
  fclose(file);
}

void printTable(double *x)
{
  int i, j;
  for (i = 0; i < I; i++) {
    printf("\t");
    for (j = 0; j < J; j++) printf("%lf ", x[i*J+j]);
  }
}

/******---------Device functions---------******/
__device__ void sort(double *dist, int n){
  int i, tmp, z;
  for(i = 1; i < n; ++i){
    tmp = dist[i];
    z = i;
    while(z > 0 && tmp < dist[z - 1]) {
      dist[z] = dist[z - 1];
      --z;
    }
    dist[z] = tmp;
  }
}

__device__ double KNN(double *X, int indexi, int n){
  int j, l, cnt = 1;                         								// initialize counters for every i element
  double distance, *dist;
  dist = (double *)malloc(n * sizeof(double));
  for (j = 0; j < I; j = j + 2)
  {
    distance = 0;                           								// initialize dist sum for every j
    if (j == indexi) continue;                    							// distance = 0, duh
    for (l = 0; l < J; l++)	distance += pow(X[indexi+l] - X[j+l], 2);
    distance = sqrt(distance);
    if (cnt <= n)
    {
      dist[cnt - 1] = distance;
      if (cnt == n) sort(dist, n);
	cnt++;
    }
    else
    {
      if (dist[cnt - 2] > distance)
      {
	dist[cnt - 2] = distance;
	sort(dist, n);
      }
    }
  }
  return dist[n-1];
}

__global__ void kernel(int *conv, int *k, double *X, double *Y, double *M){
  int indexi = blockIdx.x*blockDim.x + threadIdx.x * 2;							// index for every i element
  int realIndex = blockIdx.x*blockDim.x + threadIdx.x;
  if(indexi < I*J){

    int j, l, z, c = *conv, n = *k;									// iterators
    double sum1, sum2, distance, MeanshRange, bandwidth = 0, ynew[2], yprevious[2];
    MeanshRange = 1000000; sum1 = 0; sum2 = 0;

    for(z = 0; z < J; z++) ynew[z] = 0;									// reset ynew[] for every i
    bandwidth = KNN(X, indexi, n);									// bandwidth = distance between i and k neighbor

    for(z = 0; z < c; z++) {
      for(j = 0; j < I; j = j + 2) {
	sum1 = 0; distance = 0;
	for(l = 0; l < J; l++) distance += pow(Y[indexi+l] - X[j+l], 2);
	distance = sqrt(distance);
	if (distance <= bandwidth);
	{
	  sum1 = gaussian(distance, bandwidth);
	  sum2 += gaussian(distance, bandwidth);
	  for(l = 0; l < J; l++) ynew[l] += sum1 * X[j+l];	     
	}
      }
      MeanshRange = 0;
      for(l = 0; l < J; l++) {
	yprevious[l] = Y[indexi+l];
	ynew[l] = ynew[l] / sum2;
	Y[indexi+l] = ynew[l];
	MeanshRange += pow(ynew[l] - yprevious[l], 2);
      }
      MeanshRange = sqrt(MeanshRange);
    }

    for(l = 0; l < J; l++) M[indexi+l] = ynew[l];
  }
}
